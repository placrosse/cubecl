#include "hip/hip_runtime.h"

[START_KERNEL_COMPILATION]
name: cubecl_linalg::matmul::components::batch::base::launch::Launch<
    f32,
    cubecl_linalg::matmul::components::batch::one_to_one::Matmul<
        f32,
        half::binary16::f16,
        cubecl_linalg::matmul::components::global::homogeneous::base::Matmul<
            f32,
            half::binary16::f16,
            cubecl_linalg::matmul::components::stage::multi_buffer::base::Matmul<
                half::binary16::f16,
                f32,
                f32,
                cubecl_linalg::matmul::components::tile::accelerated::Accelerated16x16x16<
                    half::binary16::f16,
                    f32,
                >,
                cubecl_linalg::matmul::components::stage::base::S4x4x2,
            >,
        >,
        cubecl_linalg::matmul::components::batch::cube_dispatch::NaturalDispatch,
    >,
    cubecl_cuda::runtime::CudaRuntime,
>
cube_dim: (32, 4, 1)
shared_memory: 12288 bytes
info: KernelId {
    type_id: TypeId {
        t: (
            1277700598496077665,
            13781536268853219741,
        ),
    },
    info: Some (
         (
            CubeDim {
                x: 32,
                y: 4,
                z: 1,
            },
            Config {
                gmm_config: Config {
                    smm_config: Config {
                        tmm_config: Config {
                            plane_dim: 32,
                            lhs_layout: RowMajor,
                            rhs_layout: RowMajor,
                            lhs_line_size: 8,
                            rhs_line_size: 8,
                            out_line_size: 8,
                        },
                        lhs_stage_dim: StageDim {
                            tile_size_x: 16,
                            tile_size_y: 16,
                            num_tiles_x: 4,
                            num_tiles_y: 2,
                            num_tiles_per_buffer: 4,
                        },
                        rhs_stage_dim: StageDim {
                            tile_size_x: 16,
                            tile_size_y: 16,
                            num_tiles_x: 2,
                            num_tiles_y: 4,
                            num_tiles_per_buffer: 4,
                        },
                        out_stage_dim: StageDim {
                            tile_size_x: 16,
                            tile_size_y: 16,
                            num_tiles_x: 4,
                            num_tiles_y: 4,
                            num_tiles_per_buffer: 0,
                        },
                        num_planes: 4,
                        lhs_tiling_order: ColMajor,
                        rhs_tiling_order: RowMajor,
                    },
                    check_m_bounds: false,
                    check_n_bounds: false,
                    lhs_layout: RowMajor,
                    rhs_layout: RowMajor,
                    lhs_line_size: 8,
                    rhs_line_size: 8,
                    out_line_size: 8,
                },
                cube_count: (
                    2,
                    2,
                    3,
                ),
                _c: PhantomData<cubecl_linalg: : matmul: : components: : batch: : cube_dispatch: : NaturalDispatch>,
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
        ),
    ),
    mode: Some (
        Unchecked,
    ),
}
source:
```cpp
#include <mma.h>
#include <hip/hip_fp16.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

struct __align__(16) __half2_4 {
    __half2 i_0;
    __half2 i_1;
    __half2 i_2;
    __half2 i_3;
};

struct __align__(32) float_8 {
    float i_0;
    float i_1;
    float i_2;
    float i_3;
    float i_4;
    float i_5;
    float i_6;
    float i_7;
};

struct __align__(16) __half_8 {
    __half i_0;
    __half i_1;
    __half i_2;
    __half i_3;
    __half i_4;
    __half i_5;
    __half i_6;
    __half i_7;
};


extern "C" __global__ void kernel(
float_8 input_0[],float_8 input_1[],float_8 output_0[],uint info[]
) {
__shared__ __half_8 shared_memory_0[256];
__shared__ __half_8 shared_memory_1[256];
__shared__ float_8 shared_memory_2[128];
uint l_0_0;
uint l_0_1;
uint l_0_2;
uint l_0_3;
uint l_0_4;
uint l_0_5;
uint l_0_6;
uint l_0_7;
uint l_0_8;
uint l_0_9;
uint l_0_10;
uint l_0_11;
uint l_0_12;
uint l_0_13;
uint l_0_14;
uint l_0_15;
uint l_0_16;
uint l_0_17;
uint l_0_18;
uint l_0_19;
uint l_0_20;
uint l_0_21;
uint l_0_22;
uint l_0_23;
uint l_0_24;
uint l_0_25;
uint l_0_26;
uint l_0_27;
uint l_0_28;
uint l_0_29;
uint l_0_30;
uint l_0_31;
uint l_0_32;
uint l_0_33;
uint l_0_34;
uint l_0_35;
uint l_0_36;
uint l_0_37;
uint l_0_38;
bool l_0_39;
bool l_0_40;
float_8 l_0_41;
float_8 l_0_42;
float_8 l_0_43;
__half_8 l_0_44;
nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_0_0;
nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_1_0;
nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_2_0;
nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_3_0;
l_0_0 = blockIdx.x * uint(64);
l_0_1 = blockIdx.y * uint(64);
l_0_2 = info[uint(6)];
l_0_2 = l_0_2 - uint(1);
l_0_3 = info[info[uint(9)] + l_0_2];
nvcuda::wmma::fill_fragment(frag_0_0, float(0.0));
l_0_2 = uint(16);
nvcuda::wmma::fill_fragment(frag_1_0, float(0.0));
l_0_4 = uint(16);
nvcuda::wmma::fill_fragment(frag_2_0, float(0.0));
l_0_5 = uint(16);
nvcuda::wmma::fill_fragment(frag_3_0, float(0.0));
l_0_6 = uint(16);
l_0_7 = info[uint(6)];
l_0_8 = l_0_7 - uint(2);
l_0_9 = info[info[uint(12)] + l_0_8];
l_0_8 = l_0_7 - uint(1);
l_0_10 = info[info[uint(12)] + l_0_8];
l_0_8 = l_0_7 - uint(2);
l_0_11 = info[info[uint(9)] + l_0_8];
l_0_8 = l_0_7 - uint(1);
l_0_12 = info[info[uint(9)] + l_0_8];
l_0_8 = l_0_7 - uint(3);
l_0_7 = info[info[uint(12)] + l_0_8];
l_0_8 = l_0_0;
l_0_13 = uint(0);
l_0_7 = blockIdx.z * l_0_7;
l_0_14 = info[uint(7)];
l_0_15 = l_0_14 - uint(2);
l_0_16 = info[info[uint(13)] + l_0_15];
l_0_15 = l_0_14 - uint(1);
l_0_17 = info[info[uint(13)] + l_0_15];
l_0_15 = l_0_14 - uint(2);
l_0_18 = info[info[uint(10)] + l_0_15];
l_0_15 = l_0_14 - uint(1);
l_0_19 = info[info[uint(10)] + l_0_15];
l_0_15 = l_0_14 - uint(3);
l_0_14 = info[info[uint(13)] + l_0_15];
l_0_15 = uint(0);
l_0_20 = l_0_1;
l_0_14 = blockIdx.z * l_0_14;
l_0_21 = info[uint(8)];
l_0_22 = l_0_21 - uint(2);
l_0_23 = info[info[uint(14)] + l_0_22];
l_0_22 = l_0_21 - uint(1);
l_0_24 = info[info[uint(14)] + l_0_22];
l_0_22 = l_0_21 - uint(2);
l_0_25 = info[info[uint(11)] + l_0_22];
l_0_22 = l_0_21 - uint(1);
l_0_26 = info[info[uint(11)] + l_0_22];
l_0_22 = l_0_21 - uint(3);
l_0_21 = info[info[uint(14)] + l_0_22];
l_0_22 = blockIdx.z * l_0_21;
l_0_21 = l_0_3 - uint(0);
l_0_21 = l_0_21 + uint(32);
l_0_21 = l_0_21 - uint(1);
l_0_21 = l_0_21 / uint(32);

for (uint l_1_0 = uint(0); l_1_0 < l_0_21; ++l_1_0) {
nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, __half, nvcuda::wmma::row_major> frag_0_1;
nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, __half, nvcuda::wmma::row_major> frag_1_1;
l_0_27 = threadIdx.y * uint(32);
l_0_27 = l_0_27 + threadIdx.x;
l_0_27 = l_0_27 * uint(8);

for (uint l_2_0 = uint(0); l_2_0 < uint(2); ++l_2_0) {
l_0_28 = l_2_0 * uint(1024);
l_0_28 = l_0_27 + l_0_28;
l_0_29 = l_0_28 / uint(256);
l_0_30 = l_0_28 % uint(256);
l_0_31 = l_0_29 % uint(4);
l_0_32 = l_0_29 / uint(4);
l_0_33 = l_0_31 * uint(16);
l_0_33 = l_0_33 + l_0_8;
l_0_34 = l_0_32 * uint(16);
l_0_34 = l_0_34 + l_0_13;
l_0_35 = l_0_30 / uint(16);
l_0_36 = l_0_30 % uint(16);
l_0_33 = l_0_33 + l_0_35;
l_0_34 = l_0_34 + l_0_36;
l_0_37 = l_0_33 * l_0_9;
l_0_38 = l_0_34 * l_0_10;
l_0_38 = l_0_37 + l_0_38;
l_0_38 = l_0_38 + l_0_7;
l_0_38 = l_0_38 / uint(8);
l_0_39 = l_0_33 < l_0_11;
l_0_40 = l_0_34 < l_0_12;
l_0_40 = l_0_39 && l_0_40;
l_0_41 = input_0[l_0_38];
l_0_43 = float_8{
float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),};
l_0_43 = float_8 {
(l_0_40) ? l_0_41.i_0 : l_0_43.i_0,
(l_0_40) ? l_0_41.i_1 : l_0_43.i_1,
(l_0_40) ? l_0_41.i_2 : l_0_43.i_2,
(l_0_40) ? l_0_41.i_3 : l_0_43.i_3,
(l_0_40) ? l_0_41.i_4 : l_0_43.i_4,
(l_0_40) ? l_0_41.i_5 : l_0_43.i_5,
(l_0_40) ? l_0_41.i_6 : l_0_43.i_6,
(l_0_40) ? l_0_41.i_7 : l_0_43.i_7,
};
l_0_38 = l_0_28 / uint(8);
l_0_44 = __half_8{
__half(l_0_43.i_0),__half(l_0_43.i_1),__half(l_0_43.i_2),__half(l_0_43.i_3),__half(l_0_43.i_4),__half(l_0_43.i_5),__half(l_0_43.i_6),__half(l_0_43.i_7),};
shared_memory_0[l_0_38] = l_0_44;
}
l_0_38 = threadIdx.y * uint(32);
l_0_38 = l_0_38 + threadIdx.x;
l_0_38 = l_0_38 * uint(8);

for (uint l_2_0 = uint(0); l_2_0 < uint(2); ++l_2_0) {
l_0_37 = l_2_0 * uint(1024);
l_0_37 = l_0_38 + l_0_37;
l_0_36 = l_0_37 / uint(256);
l_0_35 = l_0_37 % uint(256);
l_0_34 = l_0_36 / uint(4);
l_0_33 = l_0_36 % uint(4);
l_0_32 = l_0_34 * uint(16);
l_0_32 = l_0_32 + l_0_15;
l_0_31 = l_0_33 * uint(16);
l_0_31 = l_0_31 + l_0_20;
l_0_30 = l_0_35 / uint(16);
l_0_29 = l_0_35 % uint(16);
l_0_32 = l_0_32 + l_0_30;
l_0_31 = l_0_31 + l_0_29;
l_0_28 = l_0_32 * l_0_16;
l_0_27 = l_0_31 * l_0_17;
l_0_28 = l_0_28 + l_0_27;
l_0_28 = l_0_28 + l_0_14;
l_0_28 = l_0_28 / uint(8);
l_0_40 = l_0_32 < l_0_18;
l_0_39 = l_0_31 < l_0_19;
l_0_40 = l_0_40 && l_0_39;
l_0_43 = input_1[l_0_28];
l_0_41 = float_8{
float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),float(0.0),};
l_0_43 = float_8 {
(l_0_40) ? l_0_43.i_0 : l_0_41.i_0,
(l_0_40) ? l_0_43.i_1 : l_0_41.i_1,
(l_0_40) ? l_0_43.i_2 : l_0_41.i_2,
(l_0_40) ? l_0_43.i_3 : l_0_41.i_3,
(l_0_40) ? l_0_43.i_4 : l_0_41.i_4,
(l_0_40) ? l_0_43.i_5 : l_0_41.i_5,
(l_0_40) ? l_0_43.i_6 : l_0_41.i_6,
(l_0_40) ? l_0_43.i_7 : l_0_41.i_7,
};
l_0_37 = l_0_37 / uint(8);
l_0_44 = __half_8{
__half(l_0_43.i_0),__half(l_0_43.i_1),__half(l_0_43.i_2),__half(l_0_43.i_3),__half(l_0_43.i_4),__half(l_0_43.i_5),__half(l_0_43.i_6),__half(l_0_43.i_7),};
shared_memory_1[l_0_37] = l_0_44;
}
__syncthreads();
l_0_38 = uint(16);
l_0_37 = uint(16);
l_0_36 = uint(0) * uint(4);
l_0_36 = l_0_36 + threadIdx.y;
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_0_length = l_0_35 - l_0_36;
__half_8 *slice_1_0 = shared_memory_0 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_0_1, reinterpret_cast<__half *>(slice_1_0), l_0_38);
l_0_36 = uint(0) * uint(4);
l_0_36 = l_0_36 + uint(0);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_1_length = l_0_35 - l_0_36;
__half_8 *slice_1_1 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_1), l_0_37);
nvcuda::wmma::mma_sync(frag_0_0, frag_0_1, frag_1_1, frag_0_0);
l_0_36 = uint(0) * uint(4);
l_0_36 = l_0_36 + uint(1);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_2_length = l_0_35 - l_0_36;
__half_8 *slice_1_2 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_2), l_0_37);
nvcuda::wmma::mma_sync(frag_1_0, frag_0_1, frag_1_1, frag_1_0);
l_0_36 = uint(0) * uint(4);
l_0_36 = l_0_36 + uint(2);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_3_length = l_0_35 - l_0_36;
__half_8 *slice_1_3 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_3), l_0_37);
nvcuda::wmma::mma_sync(frag_2_0, frag_0_1, frag_1_1, frag_2_0);
l_0_36 = uint(0) * uint(4);
l_0_36 = l_0_36 + uint(3);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_4_length = l_0_35 - l_0_36;
__half_8 *slice_1_4 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_4), l_0_37);
nvcuda::wmma::mma_sync(frag_3_0, frag_0_1, frag_1_1, frag_3_0);
l_0_36 = uint(1) * uint(4);
l_0_36 = l_0_36 + threadIdx.y;
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_5_length = l_0_35 - l_0_36;
__half_8 *slice_1_5 = shared_memory_0 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_0_1, reinterpret_cast<__half *>(slice_1_5), l_0_38);
l_0_36 = uint(1) * uint(4);
l_0_36 = l_0_36 + uint(0);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_6_length = l_0_35 - l_0_36;
__half_8 *slice_1_6 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_6), l_0_37);
nvcuda::wmma::mma_sync(frag_0_0, frag_0_1, frag_1_1, frag_0_0);
l_0_36 = uint(1) * uint(4);
l_0_36 = l_0_36 + uint(1);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_7_length = l_0_35 - l_0_36;
__half_8 *slice_1_7 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_7), l_0_37);
nvcuda::wmma::mma_sync(frag_1_0, frag_0_1, frag_1_1, frag_1_0);
l_0_36 = uint(1) * uint(4);
l_0_36 = l_0_36 + uint(2);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_8_length = l_0_35 - l_0_36;
__half_8 *slice_1_8 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_8), l_0_37);
nvcuda::wmma::mma_sync(frag_2_0, frag_0_1, frag_1_1, frag_2_0);
l_0_36 = uint(1) * uint(4);
l_0_36 = l_0_36 + uint(3);
l_0_36 = l_0_36 * uint(32);
l_0_35 = l_0_36 + uint(32);
const uint slice_1_9_length = l_0_35 - l_0_36;
__half_8 *slice_1_9 = shared_memory_1 + l_0_36;
nvcuda::wmma::load_matrix_sync(frag_1_1, reinterpret_cast<__half *>(slice_1_9), l_0_37);
nvcuda::wmma::mma_sync(frag_3_0, frag_0_1, frag_1_1, frag_3_0);
__syncthreads();
l_0_13 = l_0_13 + uint(32);
l_0_15 = l_0_15 + uint(32);
}
l_0_38 = uint(32) * threadIdx.y;
l_0_37 = l_0_38 + uint(32);
const uint slice_0_0_length = l_0_37 - l_0_38;
float_8 *slice_0_0 = shared_memory_2 + l_0_38;
nvcuda::wmma::store_matrix_sync(reinterpret_cast<float *>(slice_0_0), frag_0_0, l_0_2, nvcuda::wmma::mem_row_major);

for (uint l_1_0 = uint(0); l_1_0 < uint(1); ++l_1_0) {
l_0_37 = threadIdx.x * uint(8);
l_0_36 = l_1_0 * uint(256);
l_0_37 = l_0_37 + l_0_36;
l_0_36 = l_0_37 / uint(8);
l_0_43 = slice_0_0[l_0_36];
l_0_36 = threadIdx.y * uint(16);
l_0_35 = l_0_37 / uint(16);
l_0_36 = l_0_36 + l_0_35;
l_0_36 = l_0_36 + l_0_0;
l_0_35 = uint(0) * uint(16);
l_0_37 = l_0_37 % uint(16);
l_0_37 = l_0_35 + l_0_37;
l_0_37 = l_0_37 + l_0_1;
l_0_35 = l_0_36 * l_0_23;
l_0_34 = l_0_37 * l_0_24;
l_0_35 = l_0_35 + l_0_34;
l_0_35 = l_0_35 + l_0_22;
l_0_35 = l_0_35 / uint(8);
output_0[l_0_35] = l_0_43;
}
l_0_37 = l_0_38 + uint(32);
const uint slice_0_1_length = l_0_37 - l_0_38;
float_8 *slice_0_1 = shared_memory_2 + l_0_38;
nvcuda::wmma::store_matrix_sync(reinterpret_cast<float *>(slice_0_1), frag_1_0, l_0_4, nvcuda::wmma::mem_row_major);

for (uint l_1_0 = uint(0); l_1_0 < uint(1); ++l_1_0) {
l_0_37 = threadIdx.x * uint(8);
l_0_36 = l_1_0 * uint(256);
l_0_37 = l_0_37 + l_0_36;
l_0_36 = l_0_37 / uint(8);
l_0_43 = slice_0_1[l_0_36];
l_0_36 = threadIdx.y * uint(16);
l_0_35 = l_0_37 / uint(16);
l_0_36 = l_0_36 + l_0_35;
l_0_36 = l_0_36 + l_0_0;
l_0_35 = uint(1) * uint(16);
l_0_37 = l_0_37 % uint(16);
l_0_37 = l_0_35 + l_0_37;
l_0_37 = l_0_37 + l_0_1;
l_0_35 = l_0_36 * l_0_23;
l_0_34 = l_0_37 * l_0_24;
l_0_35 = l_0_35 + l_0_34;
l_0_35 = l_0_35 + l_0_22;
l_0_35 = l_0_35 / uint(8);
output_0[l_0_35] = l_0_43;
}
l_0_37 = l_0_38 + uint(32);
const uint slice_0_2_length = l_0_37 - l_0_38;
float_8 *slice_0_2 = shared_memory_2 + l_0_38;
nvcuda::wmma::store_matrix_sync(reinterpret_cast<float *>(slice_0_2), frag_2_0, l_0_5, nvcuda::wmma::mem_row_major);

for (uint l_1_0 = uint(0); l_1_0 < uint(1); ++l_1_0) {
l_0_37 = threadIdx.x * uint(8);
l_0_36 = l_1_0 * uint(256);
l_0_37 = l_0_37 + l_0_36;
l_0_36 = l_0_37 / uint(8);
l_0_43 = slice_0_2[l_0_36];
l_0_36 = threadIdx.y * uint(16);
l_0_35 = l_0_37 / uint(16);
l_0_36 = l_0_36 + l_0_35;
l_0_36 = l_0_36 + l_0_0;
l_0_35 = uint(2) * uint(16);
l_0_37 = l_0_37 % uint(16);
l_0_37 = l_0_35 + l_0_37;
l_0_37 = l_0_37 + l_0_1;
l_0_35 = l_0_36 * l_0_23;
l_0_34 = l_0_37 * l_0_24;
l_0_35 = l_0_35 + l_0_34;
l_0_35 = l_0_35 + l_0_22;
l_0_35 = l_0_35 / uint(8);
output_0[l_0_35] = l_0_43;
}
l_0_37 = l_0_38 + uint(32);
const uint slice_0_3_length = l_0_37 - l_0_38;
float_8 *slice_0_3 = shared_memory_2 + l_0_38;
nvcuda::wmma::store_matrix_sync(reinterpret_cast<float *>(slice_0_3), frag_3_0, l_0_6, nvcuda::wmma::mem_row_major);

for (uint l_1_0 = uint(0); l_1_0 < uint(1); ++l_1_0) {
l_0_37 = threadIdx.x * uint(8);
l_0_36 = l_1_0 * uint(256);
l_0_37 = l_0_37 + l_0_36;
l_0_36 = l_0_37 / uint(8);
l_0_43 = slice_0_3[l_0_36];
l_0_36 = threadIdx.y * uint(16);
l_0_35 = l_0_37 / uint(16);
l_0_36 = l_0_36 + l_0_35;
l_0_36 = l_0_36 + l_0_0;
l_0_35 = uint(3) * uint(16);
l_0_37 = l_0_37 % uint(16);
l_0_37 = l_0_35 + l_0_37;
l_0_37 = l_0_37 + l_0_1;
l_0_35 = l_0_36 * l_0_23;
l_0_34 = l_0_37 * l_0_24;
l_0_35 = l_0_35 + l_0_34;
l_0_35 = l_0_35 + l_0_22;
l_0_35 = l_0_35 / uint(8);
output_0[l_0_35] = l_0_43;
}

}
```
[END_KERNEL_COMPILATION]

