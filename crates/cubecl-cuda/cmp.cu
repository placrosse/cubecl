#include "hip/hip_runtime.h"

[START_KERNEL_COMPILATION]
name: cubecl_linalg::matmul::kernels::cmma_old::base::cmma_launch::CmmaLaunch<
    f32,
    half::binary16::f16,
    cubecl_cuda::runtime::CudaRuntime,
>
cube_dim: (32, 4, 1)
shared_memory: 12288 bytes
info: KernelId {
    type_id: TypeId {
        t: (
            11245507605303491359,
            11119494559033075139,
        ),
    },
    info: Some (
         (
            CubeDim {
                x: 32,
                y: 4,
                z: 1,
            },
            ComptimeCmmaInfo {
                block_size_m: 64,
                block_size_k: 32,
                block_size_n: 64,
                tile_size_m: 16,
                tile_size_k: 16,
                tile_size_n: 16,
                check_m_bounds: false,
                check_k_bounds: false,
                check_n_bounds: false,
                unroll: false,
                plane_dim: 32,
                num_compute_planes: 4,
                num_buffers: 2,
                num_accumulators: 4,
                write_out_strategy: ReuseSmem,
                rasterization_strategy: Swizzle,
                compute_loop_order_strategy: AllAccumulatorsFirst (
                    true,
                ),
                lhs_smem_loader_strategy: Continuous (
                    RowMajor,
                ),
                rhs_smem_loader_strategy: Continuous (
                    RowMajor,
                ),
                main_loop_strategy: Standard,
                num_compute_planes_strategy: NumTilesM,
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
            TensorCompilationArg {
                inplace: None,
                vectorisation: Some (
                    8,
                ),
            },
        ),
    ),
    mode: Some (
        Unchecked,
    ),
}
source:
```cpp
#include <mma.h>
#include <hip/hip_fp16.h>
typedef unsigned char uint8;
typedef unsigned short uint16;
typedef unsigned int uint;
typedef unsigned long long int uint64;
typedef long long int int64;

struct __align__(32) float_8 {
    float i_0;
    float i_1;
    float i_2;
    float i_3;
    float i_4;
    float i_5;
    float i_6;
    float i_7;
};


extern "C" __global__ void kernel(
float_8 input_0[],float_8 input_1[],float_8 output_0[],uint info[]
) {
    __shared__ __half shared_memory_0[2048];
    __shared__ __half shared_memory_1[2048];
    __shared__ float shared_memory_2[1024];
    uint l_0_0;
    uint l_0_1;
    uint l_0_2;
    uint l_0_3;
    uint l_0_4;
    uint l_0_5;
    uint l_0_6;
    uint l_0_7;
    uint l_0_8;
    uint l_0_9;
    uint l_0_10;
    uint l_0_11;
    uint l_0_12;
    uint l_0_13;
    uint l_0_14;
    uint l_0_15;
    uint l_0_16;
    uint l_0_17;
    uint l_0_18;
    uint l_0_19;
    uint l_0_20;
    uint l_0_21;
    uint l_0_22;
    float_8 l_0_23;
    float l_0_24;
    __half l_0_25;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_0_0;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_1_0;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_2_0;
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, 16, 16, 16, float> frag_3_0;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, 16, 16, 16, __half, nvcuda::wmma::row_major> frag_4_0;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, 16, 16, 16, __half, nvcuda::wmma::row_major> frag_5_0;
    l_0_0 = info[uint(6)];
    l_0_1 = l_0_0 - uint(2);
    l_0_0 = l_0_0 - uint(1);
    l_0_2 = info[info[uint(9)] + l_0_1];
    l_0_1 = info[info[uint(9)] + l_0_0];
    l_0_3 = info[info[uint(10)] + l_0_0];
    l_0_0 = gridDim.y * uint(2);
    l_0_4 = blockIdx.x * gridDim.y;
    l_0_4 = l_0_4 + blockIdx.y;
    l_0_5 = l_0_4 % l_0_0;
    l_0_6 = l_0_4 / l_0_0;
    l_0_7 = l_0_5 / gridDim.y;
    l_0_6 = l_0_6 * uint(2);
    l_0_7 = l_0_6 + l_0_7;
    l_0_6 = l_0_5 % gridDim.y;
    l_0_5 = l_0_4 / l_0_0;
    l_0_5 = l_0_5 % uint(2);
    l_0_4 = uint(2) * l_0_6;
    l_0_4 = gridDim.y - l_0_4;
    l_0_4 = l_0_4 - uint(1);
    l_0_5 = l_0_5 * l_0_4;
    l_0_6 = l_0_6 + l_0_5;
    l_0_5 = l_0_6 * uint(64);
    l_0_4 = l_0_7 * uint(64);
    l_0_7 = info[uint(8)];
    l_0_6 = l_0_7 - uint(2);
    l_0_0 = info[info[uint(9)] + l_0_6];
    l_0_6 = l_0_7 - uint(1);
    l_0_8 = info[info[uint(10)] + l_0_6];
    l_0_8 = l_0_0 * l_0_8;
    l_0_8 = l_0_8 * blockIdx.z;
    l_0_6 = uint(0);
    l_0_0 = uint(0);
    l_0_7 = l_0_7 - uint(2);

    for (uint l_1_0 = uint(0); l_1_0 < l_0_7; ++l_1_0) {
        l_0_9 = info[info[uint(14)] + l_1_0];
        l_0_9 = l_0_8 / l_0_9;
        l_0_10 = info[info[uint(9)] + l_1_0];
        l_0_10 = l_0_9 % l_0_10;
        l_0_11 = info[info[uint(12)] + l_1_0];
        l_0_11 = l_0_10 * l_0_11;
        l_0_6 = l_0_6 + l_0_11;
        l_0_11 = info[info[uint(10)] + l_1_0];
        l_0_11 = l_0_9 % l_0_11;
        l_0_10 = info[info[uint(13)] + l_1_0];
        l_0_11 = l_0_11 * l_0_10;
        l_0_0 = l_0_0 + l_0_11;
    }
    l_0_11 = l_0_5;
    l_0_10 = l_0_4;
    l_0_9 = threadIdx.y;
    l_0_7 = threadIdx.x;
    l_0_5 = threadIdx.y;
    l_0_4 = threadIdx.x;
    nvcuda::wmma::fill_fragment(frag_0_0, float(0.0));
    nvcuda::wmma::fill_fragment(frag_1_0, float(0.0));
    nvcuda::wmma::fill_fragment(frag_2_0, float(0.0));
    nvcuda::wmma::fill_fragment(frag_3_0, float(0.0));
    l_0_12 = l_0_1 + uint(32);
    l_0_12 = l_0_12 - uint(1);
    l_0_12 = l_0_12 / uint(32);

    // global loop
    for (uint l_1_0 = uint(0); l_1_0 < l_0_12; ++l_1_0) {
        l_0_13 = l_1_0 * uint(32);
        l_0_14 = uint(32);
        l_0_15 = uint(64);
        l_0_15 = l_0_14 * l_0_15;
        l_0_14 = uint(4) * uint(8);
        l_0_14 = l_0_14 * uint(32);
        l_0_15 = l_0_15 / l_0_14;
        l_0_16 = l_0_5 * uint(32);
        l_0_16 = l_0_16 + l_0_4;
        l_0_16 = l_0_16 * uint(8);

        // load lhs
        for (uint l_2_0 = uint(0); l_2_0 < l_0_15; ++l_2_0) {
            l_0_17 = l_2_0 * l_0_14;
            l_0_17 = l_0_16 + l_0_17;
            l_0_18 = uint(16);
            l_0_19 = uint(16);
            l_0_19 = l_0_18 * l_0_19;
            l_0_18 = uint(32);
            l_0_20 = uint(16);
            l_0_20 = l_0_18 / l_0_20;
            l_0_18 = uint(64);
            l_0_21 = uint(16);
            l_0_21 = l_0_18 / l_0_21;
            l_0_18 = l_0_17 / l_0_19;
            l_0_22 = l_0_18 / l_0_20;
            l_0_20 = l_0_18 % l_0_20;
            l_0_21 = uint(16);
            l_0_19 = l_0_17 % l_0_19;
            l_0_18 = l_0_19 / l_0_21;
            l_0_21 = l_0_19 % l_0_21;
            l_0_19 = uint(16);
            l_0_19 = l_0_22 * l_0_19;
            l_0_19 = l_0_19 + l_0_18;
            l_0_18 = uint(16);
            l_0_18 = l_0_20 * l_0_18;
            l_0_21 = l_0_18 + l_0_21;
            l_0_22 = l_0_19 + l_0_11;
            l_0_20 = l_0_21 + l_0_13;
            l_0_22 = l_0_22 * l_0_1;
            l_0_22 = l_0_6 + l_0_22;
            l_0_22 = l_0_22 + l_0_20;
            l_0_22 = l_0_22 / uint(8);
            l_0_23 = input_0[l_0_22];
            l_0_22 = l_0_17 + uint(0);
            l_0_24 = l_0_23.i_0;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(1);
            l_0_24 = l_0_23.i_1;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(2);
            l_0_24 = l_0_23.i_2;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(3);
            l_0_24 = l_0_23.i_3;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(4);
            l_0_24 = l_0_23.i_4;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(5);
            l_0_24 = l_0_23.i_5;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(6);
            l_0_24 = l_0_23.i_6;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
            l_0_22 = l_0_17 + uint(7);
            l_0_24 = l_0_23.i_7;
            l_0_25 = __half(l_0_24);
            shared_memory_0[l_0_22] = l_0_25;
        }
        l_0_22 = uint(64);
        l_0_21 = uint(32);
        l_0_22 = l_0_22 * l_0_21;
        l_0_21 = uint(4) * uint(8);
        l_0_21 = l_0_21 * uint(32);
        l_0_22 = l_0_22 / l_0_21;
        l_0_20 = l_0_5 * uint(32);
        l_0_20 = l_0_20 + l_0_4;
        l_0_20 = l_0_20 * uint(8);

        // load rhs
        for (uint l_2_0 = uint(0); l_2_0 < l_0_22; ++l_2_0) {
            l_0_19 = l_2_0 * l_0_21;
            l_0_19 = l_0_20 + l_0_19;
            l_0_18 = uint(16);
            l_0_17 = uint(16);
            l_0_18 = l_0_18 * l_0_17;
            l_0_17 = uint(64);
            l_0_16 = uint(16);
            l_0_17 = l_0_17 / l_0_16;
            l_0_16 = uint(32);
            l_0_15 = uint(16);
            l_0_16 = l_0_16 / l_0_15;
            l_0_15 = l_0_19 / l_0_18;
            l_0_14 = l_0_15 / l_0_17;
            l_0_17 = l_0_15 % l_0_17;
            l_0_16 = uint(16);
            l_0_18 = l_0_19 % l_0_18;
            l_0_15 = l_0_18 / l_0_16;
            l_0_18 = l_0_18 % l_0_16;
            l_0_16 = uint(16);
            l_0_16 = l_0_14 * l_0_16;
            l_0_16 = l_0_16 + l_0_15;
            l_0_15 = uint(16);
            l_0_15 = l_0_17 * l_0_15;
            l_0_18 = l_0_15 + l_0_18;
            l_0_17 = l_0_16 + l_0_13;
            l_0_15 = l_0_18 + l_0_10;
            l_0_17 = l_0_17 * l_0_3;
            l_0_17 = l_0_0 + l_0_17;
            l_0_17 = l_0_17 + l_0_15;
            l_0_17 = l_0_17 / uint(8);
            l_0_23 = input_1[l_0_17];
            l_0_17 = l_0_19 + uint(0);
            l_0_24 = l_0_23.i_0;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(1);
            l_0_24 = l_0_23.i_1;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(2);
            l_0_24 = l_0_23.i_2;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(3);
            l_0_24 = l_0_23.i_3;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(4);
            l_0_24 = l_0_23.i_4;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(5);
            l_0_24 = l_0_23.i_5;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(6);
            l_0_24 = l_0_23.i_6;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
            l_0_17 = l_0_19 + uint(7);
            l_0_24 = l_0_23.i_7;
            l_0_25 = __half(l_0_24);
            shared_memory_1[l_0_17] = l_0_25;
        }
        __syncthreads();
        l_0_22 = l_0_9 / uint(1);
        l_0_21 = l_0_9 % uint(1);
        l_0_21 = l_0_21 * uint(4);

        for (uint l_2_0 = uint(0); l_2_0 < uint(2); ++l_2_0) {
            l_0_20 = uint(32);
            l_0_19 = uint(16);
            l_0_20 = l_0_20 / l_0_19;
            l_0_19 = uint(64);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 / l_0_18;
            l_0_20 = l_0_22 * l_0_20;
            l_0_20 = l_0_20 + l_2_0;
            l_0_19 = uint(16);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 * l_0_18;
            l_0_20 = l_0_20 * l_0_19;
            l_0_19 = l_0_20 + l_0_19;
            const uint slice_2_0_length = l_0_19 - l_0_20;
            __half *slice_2_0 = shared_memory_0 + l_0_20;
            l_0_20 = uint(16);
            nvcuda::wmma::load_matrix_sync(frag_4_0, slice_2_0, l_0_20);
            l_0_20 = l_0_21 + uint(0);
            l_0_19 = uint(64);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 / l_0_18;
            l_0_18 = uint(32);
            l_0_17 = uint(16);
            l_0_18 = l_0_18 / l_0_17;
            l_0_19 = l_2_0 * l_0_19;
            l_0_19 = l_0_19 + l_0_20;
            l_0_20 = uint(16);
            l_0_18 = uint(16);
            l_0_20 = l_0_20 * l_0_18;
            l_0_19 = l_0_19 * l_0_20;
            l_0_20 = l_0_19 + l_0_20;
            const uint slice_2_1_length = l_0_20 - l_0_19;
            __half *slice_2_1 = shared_memory_1 + l_0_19;
            l_0_20 = uint(16);
            nvcuda::wmma::load_matrix_sync(frag_5_0, slice_2_1, l_0_20);
            nvcuda::wmma::mma_sync(frag_0_0, frag_4_0, frag_5_0, frag_0_0);
            l_0_20 = l_0_21 + uint(1);
            l_0_19 = uint(64);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 / l_0_18;
            l_0_18 = uint(32);
            l_0_17 = uint(16);
            l_0_18 = l_0_18 / l_0_17;
            l_0_19 = l_2_0 * l_0_19;
            l_0_19 = l_0_19 + l_0_20;
            l_0_20 = uint(16);
            l_0_18 = uint(16);
            l_0_20 = l_0_20 * l_0_18;
            l_0_19 = l_0_19 * l_0_20;
            l_0_20 = l_0_19 + l_0_20;
            const uint slice_2_2_length = l_0_20 - l_0_19;
            __half *slice_2_2 = shared_memory_1 + l_0_19;
            l_0_20 = uint(16);
            nvcuda::wmma::load_matrix_sync(frag_5_0, slice_2_2, l_0_20);
            nvcuda::wmma::mma_sync(frag_1_0, frag_4_0, frag_5_0, frag_1_0);
            l_0_20 = l_0_21 + uint(2);
            l_0_19 = uint(64);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 / l_0_18;
            l_0_18 = uint(32);
            l_0_17 = uint(16);
            l_0_18 = l_0_18 / l_0_17;
            l_0_19 = l_2_0 * l_0_19;
            l_0_19 = l_0_19 + l_0_20;
            l_0_20 = uint(16);
            l_0_18 = uint(16);
            l_0_20 = l_0_20 * l_0_18;
            l_0_19 = l_0_19 * l_0_20;
            l_0_20 = l_0_19 + l_0_20;
            const uint slice_2_3_length = l_0_20 - l_0_19;
            __half *slice_2_3 = shared_memory_1 + l_0_19;
            l_0_20 = uint(16);
            nvcuda::wmma::load_matrix_sync(frag_5_0, slice_2_3, l_0_20);
            nvcuda::wmma::mma_sync(frag_2_0, frag_4_0, frag_5_0, frag_2_0);
            l_0_20 = l_0_21 + uint(3);
            l_0_19 = uint(64);
            l_0_18 = uint(16);
            l_0_19 = l_0_19 / l_0_18;
            l_0_18 = uint(32);
            l_0_17 = uint(16);
            l_0_18 = l_0_18 / l_0_17;
            l_0_19 = l_2_0 * l_0_19;
            l_0_19 = l_0_19 + l_0_20;
            l_0_20 = uint(16);
            l_0_18 = uint(16);
            l_0_20 = l_0_20 * l_0_18;
            l_0_19 = l_0_19 * l_0_20;
            l_0_20 = l_0_19 + l_0_20;
            const uint slice_2_4_length = l_0_20 - l_0_19;
            __half *slice_2_4 = shared_memory_1 + l_0_19;
            l_0_20 = uint(16);
            nvcuda::wmma::load_matrix_sync(frag_5_0, slice_2_4, l_0_20);
            nvcuda::wmma::mma_sync(frag_3_0, frag_4_0, frag_5_0, frag_3_0);
        }
        __syncthreads();
    }
    l_0_22 = l_0_9 * uint(256);
    l_0_21 = l_0_22 + uint(256);
    const uint slice_0_0_length = l_0_21 - l_0_22;
    float *slice_0_0 = shared_memory_2 + l_0_22;
    nvcuda::wmma::store_matrix_sync(slice_0_0, frag_0_0, uint(16), nvcuda::wmma::mem_row_major);
    l_0_22 = l_0_9 / uint(1);
    l_0_21 = l_0_9 % uint(1);
    l_0_21 = l_0_21 * uint(4);
    l_0_20 = l_0_9 * uint(256);
    l_0_19 = l_0_7 * uint(8);
    l_0_20 = l_0_20 + l_0_19;
    l_0_19 = l_0_7 / uint(2);
    l_0_18 = l_0_7 % uint(2);
    l_0_18 = l_0_18 * uint(8);
    l_0_22 = l_0_22 * uint(16);
    l_0_22 = l_0_11 + l_0_22;
    l_0_21 = l_0_21 * uint(16);
    l_0_21 = l_0_10 + l_0_21;
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(0) * uint(16);
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(0) * uint(256);
    l_0_18 = l_0_20 + l_0_18;
    l_0_17 = l_0_22 + l_0_19;
    l_0_16 = uint(0) * uint(16);
    l_0_17 = l_0_17 + l_0_16;
    l_0_17 = l_0_17 * l_0_3;
    l_0_17 = l_0_8 + l_0_17;
    l_0_17 = l_0_17 + l_0_21;
    l_0_16 = l_0_18 + uint(0);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_0 = l_0_24;
    l_0_16 = l_0_18 + uint(1);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_1 = l_0_24;
    l_0_16 = l_0_18 + uint(2);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_2 = l_0_24;
    l_0_16 = l_0_18 + uint(3);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_3 = l_0_24;
    l_0_16 = l_0_18 + uint(4);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_4 = l_0_24;
    l_0_16 = l_0_18 + uint(5);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_5 = l_0_24;
    l_0_16 = l_0_18 + uint(6);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_6 = l_0_24;
    l_0_16 = l_0_18 + uint(7);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_7 = l_0_24;
    l_0_17 = l_0_17 / uint(8);
    output_0[l_0_17] = l_0_23;
    nvcuda::wmma::store_matrix_sync(slice_0_0, frag_1_0, uint(16), nvcuda::wmma::mem_row_major);
    l_0_22 = l_0_9 / uint(1);
    l_0_21 = l_0_9 % uint(1);
    l_0_21 = l_0_21 * uint(4);
    l_0_20 = l_0_9 * uint(256);
    l_0_19 = l_0_7 * uint(8);
    l_0_20 = l_0_20 + l_0_19;
    l_0_19 = l_0_7 / uint(2);
    l_0_18 = l_0_7 % uint(2);
    l_0_18 = l_0_18 * uint(8);
    l_0_22 = l_0_22 * uint(16);
    l_0_22 = l_0_11 + l_0_22;
    l_0_21 = l_0_21 * uint(16);
    l_0_21 = l_0_10 + l_0_21;
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(1) * uint(16);
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(0) * uint(256);
    l_0_18 = l_0_20 + l_0_18;
    l_0_17 = l_0_22 + l_0_19;
    l_0_16 = uint(0) * uint(16);
    l_0_17 = l_0_17 + l_0_16;
    l_0_17 = l_0_17 * l_0_3;
    l_0_17 = l_0_8 + l_0_17;
    l_0_17 = l_0_17 + l_0_21;
    l_0_16 = l_0_18 + uint(0);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_0 = l_0_24;
    l_0_16 = l_0_18 + uint(1);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_1 = l_0_24;
    l_0_16 = l_0_18 + uint(2);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_2 = l_0_24;
    l_0_16 = l_0_18 + uint(3);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_3 = l_0_24;
    l_0_16 = l_0_18 + uint(4);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_4 = l_0_24;
    l_0_16 = l_0_18 + uint(5);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_5 = l_0_24;
    l_0_16 = l_0_18 + uint(6);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_6 = l_0_24;
    l_0_16 = l_0_18 + uint(7);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_7 = l_0_24;
    l_0_17 = l_0_17 / uint(8);
    output_0[l_0_17] = l_0_23;
    nvcuda::wmma::store_matrix_sync(slice_0_0, frag_2_0, uint(16), nvcuda::wmma::mem_row_major);
    l_0_22 = l_0_9 / uint(1);
    l_0_21 = l_0_9 % uint(1);
    l_0_21 = l_0_21 * uint(4);
    l_0_20 = l_0_9 * uint(256);
    l_0_19 = l_0_7 * uint(8);
    l_0_20 = l_0_20 + l_0_19;
    l_0_19 = l_0_7 / uint(2);
    l_0_18 = l_0_7 % uint(2);
    l_0_18 = l_0_18 * uint(8);
    l_0_22 = l_0_22 * uint(16);
    l_0_22 = l_0_11 + l_0_22;
    l_0_21 = l_0_21 * uint(16);
    l_0_21 = l_0_10 + l_0_21;
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(2) * uint(16);
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(0) * uint(256);
    l_0_18 = l_0_20 + l_0_18;
    l_0_17 = l_0_22 + l_0_19;
    l_0_16 = uint(0) * uint(16);
    l_0_17 = l_0_17 + l_0_16;
    l_0_17 = l_0_17 * l_0_3;
    l_0_17 = l_0_8 + l_0_17;
    l_0_17 = l_0_17 + l_0_21;
    l_0_16 = l_0_18 + uint(0);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_0 = l_0_24;
    l_0_16 = l_0_18 + uint(1);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_1 = l_0_24;
    l_0_16 = l_0_18 + uint(2);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_2 = l_0_24;
    l_0_16 = l_0_18 + uint(3);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_3 = l_0_24;
    l_0_16 = l_0_18 + uint(4);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_4 = l_0_24;
    l_0_16 = l_0_18 + uint(5);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_5 = l_0_24;
    l_0_16 = l_0_18 + uint(6);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_6 = l_0_24;
    l_0_16 = l_0_18 + uint(7);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_7 = l_0_24;
    l_0_17 = l_0_17 / uint(8);
    output_0[l_0_17] = l_0_23;
    nvcuda::wmma::store_matrix_sync(slice_0_0, frag_3_0, uint(16), nvcuda::wmma::mem_row_major);
    l_0_22 = l_0_9 / uint(1);
    l_0_21 = l_0_9 % uint(1);
    l_0_21 = l_0_21 * uint(4);
    l_0_20 = l_0_9 * uint(256);
    l_0_19 = l_0_7 * uint(8);
    l_0_20 = l_0_20 + l_0_19;
    l_0_19 = l_0_7 / uint(2);
    l_0_18 = l_0_7 % uint(2);
    l_0_18 = l_0_18 * uint(8);
    l_0_22 = l_0_22 * uint(16);
    l_0_22 = l_0_11 + l_0_22;
    l_0_21 = l_0_21 * uint(16);
    l_0_21 = l_0_10 + l_0_21;
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(3) * uint(16);
    l_0_21 = l_0_21 + l_0_18;
    l_0_18 = uint(0) * uint(256);
    l_0_18 = l_0_20 + l_0_18;
    l_0_17 = l_0_22 + l_0_19;
    l_0_16 = uint(0) * uint(16);
    l_0_17 = l_0_17 + l_0_16;
    l_0_17 = l_0_17 * l_0_3;
    l_0_17 = l_0_8 + l_0_17;
    l_0_17 = l_0_17 + l_0_21;
    l_0_16 = l_0_18 + uint(0);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_0 = l_0_24;
    l_0_16 = l_0_18 + uint(1);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_1 = l_0_24;
    l_0_16 = l_0_18 + uint(2);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_2 = l_0_24;
    l_0_16 = l_0_18 + uint(3);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_3 = l_0_24;
    l_0_16 = l_0_18 + uint(4);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_4 = l_0_24;
    l_0_16 = l_0_18 + uint(5);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_5 = l_0_24;
    l_0_16 = l_0_18 + uint(6);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_6 = l_0_24;
    l_0_16 = l_0_18 + uint(7);
    l_0_24 = shared_memory_2[l_0_16];
    l_0_23.i_7 = l_0_24;
    l_0_17 = l_0_17 / uint(8);
    output_0[l_0_17] = l_0_23;

}
```
[END_KERNEL_COMPILATION]

